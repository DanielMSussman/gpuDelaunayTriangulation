#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "hiprand/hiprand_kernel.h"
#include "noiseSource.cuh"
#include "std_include.h"

/** \file noiseSource.cu
    * Defines kernel callers and kernels for GPU random number generation
*/

/*!
    \addtogroup utilityKernels
    @{
*/

/*!
  Each thread -- most likely corresponding to each cell -- is initialized with a different sequence
  of the same seed of a cudaRNG
*/
__global__ void initialize_RNG_array_kernel(hiprandState *state, int N,int Timestep,int GlobalSeed)
    {
    unsigned int idx = blockIdx.x*blockDim.x + threadIdx.x;
    if (idx >=N)
        return;
    hiprand_init(GlobalSeed,idx,Timestep,&state[idx]);
    return;
    };

//!Call the kernel to initialize a different RNG for each particle
bool gpu_initialize_RNG_array(hiprandState *states,
                    int N,
                    int Timestep,
                    int GlobalSeed)
    {
    unsigned int block_size = 128;
    if (N < 128) block_size = 32;
    unsigned int nblocks  = N/block_size + 1;


    initialize_RNG_array_kernel<<<nblocks,block_size>>>(states,N,Timestep,GlobalSeed);
    HANDLE_ERROR(hipGetLastError());
    return hipSuccess;
    };
