#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "cellListGPU.cuh"
#include "indexer.h"
#include "periodicBoundaries.h"
#include <iostream>
#include <stdio.h>

#define nThreads 256

/*! \file cellListGPU.cu */

/*!
    \addtogroup cellListGPUKernels
    @{
*/

/*!
  Assign particles to bins, keep track of the number of particles per bin, etc.
  */
__global__ void gpu_compute_cell_list_kernel(double2 *d_pt,
                                              unsigned int *d_cell_sizes,
                                              int *d_idx,
                                              int Np,
                                              unsigned int Nmax,
                                              int xsize,
                                              int ysize,
                                              double boxsize,
                                              periodicBoundaries Box,
                                              Index2D ci,
                                              Index2D cli,
                                              int *d_assist
                                              )
    {
    // read in the particle that belongs to this thread
    unsigned int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx >= Np)
        return;

    double2 pos = ldgHD(&d_pt[idx]);

    int ibin = max(0,min(xsize-1,(int)floor(pos.x/boxsize)));
    int jbin = max(0,min(xsize-1,(int)floor(pos.y/boxsize)));
    int bin = ci(ibin,jbin);

    unsigned int offset = atomicAdd(&(d_cell_sizes[bin]), 1);
    if (offset <= d_assist[0]+1)
        {
        unsigned int write_pos = min(cli(offset, bin),cli.getNumElements()-1);
        d_idx[write_pos] = idx;
        }
    else
        {
        d_assist[0]=offset+1;
        d_assist[1]=1;
        };

    return;
    };


/////
//Kernel callers
///


bool gpu_compute_cell_list(double2 *d_pt,
                                  unsigned int *d_cell_sizes,
                                  int *d_idx,
                                  int Np,
                                  int &Nmax,
                                  int xsize,
                                  int ysize,
                                  double boxsize,
                                  periodicBoundaries &Box,
                                  Index2D &ci,
                                  Index2D &cli,
                                  int *d_assist
                                  )
    {
    //optimize block size later
    unsigned int block_size = nThreads;
    if (Np < nThreads) block_size = 16;
    unsigned int nblocks  = Np/block_size + 1;


    unsigned int nmax = (unsigned int) Nmax;
    gpu_compute_cell_list_kernel<<<nblocks, block_size>>>(d_pt,
                                                          d_cell_sizes,
                                                          d_idx,
                                                          Np,
                                                          nmax,
                                                          xsize,
                                                          ysize,
                                                          boxsize,
                                                          Box,
                                                          ci,
                                                          cli,
                                                          d_assist
                                                          );
    HANDLE_ERROR(hipGetLastError());
#ifdef DEBUGFLAGUP
    hipDeviceSynchronize();
#endif
    return hipSuccess;
    }

/** @} */ //end of group declaration
